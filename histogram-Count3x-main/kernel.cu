
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void histo_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins)
{
	
    /*************************************************************************/
    // INSERT KERNEL CODE HERE
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Privatized bins
    extern __shared__ unsigned int bins_s[];
    for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
         binIdx += blockDim.x) {
        bins_s[binIdx] = 0;
    }
    __syncthreads();
    // Histogram
    for (unsigned int i = tid; i < num_elements; i += blockDim.x * gridDim.x) {
        atomicAdd(&(bins_s[(unsigned int) input[i]]), 1);
    }
    __syncthreads();
    // Commit to global memory
    for (unsigned int binIdx = threadIdx.x; binIdx < num_bins;
         binIdx += blockDim.x) {
        atomicAdd(&(bins[binIdx]), bins_s[binIdx]);
    }
	
	  /*************************************************************************/
}

void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

	  /*************************************************************************/
    //INSERT CODE HERE
    dim3 blockDim(512), gridDim(30);
    histo_kernel<<<gridDim, blockDim, num_bins * sizeof(unsigned int)>>>
            (input, bins, num_elements, num_bins);

	  /*************************************************************************/

}


